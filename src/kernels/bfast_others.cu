#include "hip/hip_runtime.h"
#include "../bfast_util.cu.h"
#include "bfast_helpers.cu.h"

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 1: Generating X
//
// Output:
//   X: [k2p2][N]f32

__global__ void bfast_step_1(float *X, int k2p2, int N, float f)
{
  int gidy = blockIdx.y * blockDim.y + threadIdx.y;
  int gidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (gidy >= k2p2 || gidx >= N) {
    return;
  }

  int i = gidy;
  int j = gidx + 1;
  float val;
  if (i == 0) { val = 1.0; }
  else if (i == 1) { val = (float)j; }
  else {
    float angle = 2.0 * M_PI * (float)(i / 2) * (float)j / f;
    if (i % 2 == 0) {
      val = __sinf(angle);
    } else {
      val = __cosf(angle);
    }
  }

  X[IDX_2D(gidy, gidx, N)] = val;
}

void bfast_step_1_run(struct bfast_state *s)
{
  float *d_X = fget_dev(s,X);
  int N = s->N, k2p2 = s->k2p2;
  float freq = s->freq;

  dim3 block(16, 16, 1);
  dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(k2p2, block.y), 1);
  bfast_step_1<<<grid, block>>>(d_X, k2p2, N, freq);
}

BFAST_BEGIN_TEST(bfast_step_1_test)
  BFAST_BEGIN_INPUTS { } BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS { BFAST_VALUE_X } BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_1_run) } BFAST_END_STEPS
BFAST_END_TEST

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 3: Calculating Xinv
//
// Input:
//   Xsqr: [m][k2p2][k2p2]f32
// Output:
//   Xinv: [m][k2p2][k2p2]f32
//

__global__ void bfast_step_3(float *Xsqr, float *Xinv, int k2p2)
{
  // Grid: (m, 1, 1)
  // Block: (2*k2p2, k2p2, 1)
  // NB! Uses dynamically allocated shared memory: k2p2*2*k2p2 floats per block

  if (threadIdx.x >= 2*k2p2 || threadIdx.y >= k2p2) {
    return;
  }

  float *sqr = &Xsqr[blockIdx.x * (k2p2 * k2p2)];
  float *inv = &Xinv[blockIdx.x * (k2p2 * k2p2)];

  extern __shared__ float A[]; // [k2p2][2*k2p2]

  // Body of mat_inv map
  if (threadIdx.x < k2p2) {
    // Left half of A
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] =
      sqr[IDX_2D(threadIdx.y, threadIdx.x, k2p2)];
  } else {
    // Right half of A
    float val = threadIdx.y == (threadIdx.x - k2p2) ? 1.0 : 0.0;
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] = val;
  }
  __syncthreads();

  // guass_jordan loop and map body
  for (int i = 0; i < k2p2; i++) {
    float v1 = A[i];
    float x = A[threadIdx.x] / v1;
    float val = x;

    if (threadIdx.y < k2p2 - 1) {
      val = A[IDX_2D(threadIdx.y + 1, threadIdx.x, 2*k2p2)]
              - A[IDX_2D(threadIdx.y + 1, i, 2*k2p2)] * x;
    }
    __syncthreads();
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] = val;
    __syncthreads();
  }

  // Write back to global memory
  if (threadIdx.x < k2p2) {
    inv[IDX_2D(threadIdx.y, threadIdx.x, k2p2)] =
      A[IDX_2D(threadIdx.y, threadIdx.x + k2p2, 2*k2p2)];
  }
}

void bfast_step_3_run(struct bfast_state *s)
{
  float *d_Xsqr = fget_dev(s,Xsqr), *d_Xinv = fget_dev(s,Xinv);
  int m = s->m, k2p2 = s->k2p2;

  dim3 block(16, 8, 1); // Assumes k2p2 <= 8
  dim3 grid(m, 1, 1);
  const size_t shared_size = k2p2 * 2 * k2p2 * sizeof(float);
  bfast_step_3<<<grid, block, shared_size>>>(d_Xsqr, d_Xinv, k2p2);
}

BFAST_BEGIN_TEST(bfast_step_3_test)
  BFAST_BEGIN_INPUTS { BFAST_VALUE_Xsqr } BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS { BFAST_VALUE_Xinv } BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_3_run) } BFAST_END_STEPS
BFAST_END_TEST

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 4b: Calculating beta
//
// Input:
//   Xinv:  [m][k2p2][k2p2]f32
//   beta0: [m][k2p2]f32
// Output:
//   beta:  [m][k2p2]f32
//
// Every block performs a matrix-vector multiplication between a matrix from
// Xinv and a row from beta0. The resulting vectors are rows in the final
// [m][k2p2] matrix, beta.
//

__global__ void bfast_step_4b(float *Xinv, float *beta0, float *beta, int k2p2)
{
  // Grid: (m, 1, 1)
  // Block: (k2p2, 1, 1)

  if (threadIdx.x >= k2p2) { return; }

  float *inv = &Xinv[blockIdx.x * (k2p2 * k2p2)];
  float *vct = &beta0[blockIdx.x * k2p2];
  float accum = 0.0;

  for (int i = 0; i < k2p2; i++) {
    accum += inv[IDX_2D(threadIdx.x, i, k2p2)] * vct[i];
  }

  beta[IDX_2D(blockIdx.x, threadIdx.x, blockDim.x)] = accum;
}

void bfast_step_4b_run(struct bfast_state *s)
{
  float *d_Xinv = fget_dev(s,Xinv), *d_beta0 = fget_dev(s,beta0);
  float *d_beta = fget_dev(s,beta);
  int m = s->m, k2p2 = s->k2p2;

  dim3 block(8, 1, 1); // Assumes k2p2 <= 8
  dim3 grid(m, 1, 1);
  bfast_step_4b<<<grid, block>>>(d_Xinv, d_beta0, d_beta, k2p2);
}

BFAST_BEGIN_TEST(bfast_step_4b_test)
  BFAST_BEGIN_INPUTS { BFAST_VALUE_Xinv, BFAST_VALUE_beta0 } BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS { BFAST_VALUE_beta } BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_4b_run) } BFAST_END_STEPS
BFAST_END_TEST


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 5: Calculating Nss, y_erros, val_indss
//
// Input:
//   Y:         [m][N]f32
//   y_preds:   [m][N]f32
// Output:
//   Nss:       [m]i32
//   y_errors:  [m][N]f32
//   val_indss: [m][N]i32
//

__global__ void bfast_step_5(float *Y, float *y_preds, int *Nss,
    float *y_errors, int *val_indss, int N)
{
  // Grid: (m, 1, 1)
  // Block: (N, 1, 1)

  if (threadIdx.x >= N) { return; }

  float *y = &Y[blockIdx.x * N];
  float *y_pred = &y_preds[blockIdx.x * N];
  float *y_error = &y_errors[blockIdx.x * N];
  int *val_inds = &val_indss[blockIdx.x * N];
  int *Ns = &Nss[blockIdx.x];

  float val = y[threadIdx.x];
  float err = !isnan(val) ? val - y_pred[threadIdx.x] : NAN;

  // Partition
  extern __shared__ int num_valids[]; // N
  num_valids[threadIdx.x] = !isnan(err);
  __syncthreads();
  scaninc_block_add<int>(num_valids);
  int i = num_valids[N - 1];

  unsigned int idx;
  if (!isnan(err)) {
    idx = num_valids[threadIdx.x] - 1;
  } else {
    //float num_invalids = threadIdx.x - num_valids[threadIdx.x] + 1;
    //idx = num_invalids + i - 1;
    idx = threadIdx.x - num_valids[threadIdx.x] + i;
  }

  y_error[idx] = err;
  val_inds[idx] = threadIdx.x;
  if (threadIdx.x == 0) {
    *Ns = i;
  }
}

void bfast_step_5_run(struct bfast_state *s)
{
  int m = s->m, N = s->N;
  float *d_Y = fget_dev(s,Y), *d_y_preds = fget_dev(s,y_preds);
  int *d_Nss = iget_dev(s,Nss), *d_val_indss = iget_dev(s,val_indss);
  float *d_y_errors = fget_dev(s,y_errors);

  dim3 block(N, 1, 1);
  dim3 grid(m, 1, 1);
  const size_t shared_size = N * sizeof(int);
  bfast_step_5<<<grid, block, shared_size>>>(d_Y, d_y_preds, d_Nss, d_y_errors,
                                             d_val_indss, N);
}

BFAST_BEGIN_TEST(bfast_step_5_test)
  BFAST_BEGIN_INPUTS { BFAST_VALUE_Y, BFAST_VALUE_y_preds } BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS
  {
    BFAST_VALUE_Nss,
    BFAST_VALUE_y_errors,
    BFAST_VALUE_val_indss,
  }
  BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_5_run) } BFAST_END_STEPS
BFAST_END_TEST

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 7a: Calculating MO_fsts
//
// Input:
//    y_errors: [m][N]f32
//    nss:      [m]i32
// Output:
//    MO_fsts:  [m]f32

__global__ void bfast_step_7a(float *y_errors,
                                int *nss,
                                int  h,
                                int  N,
                              float *MO_fsts)
{
  // Grid:  (m, 1, 1)
  // Block: (h, 1, 1)

  if (h <= threadIdx.x) { return; }

  float *y_error = &y_errors[blockIdx.x * N];
  float *MO_fst  = &MO_fsts [blockIdx.x];
  int    ns      = nss      [blockIdx.x];
  extern __shared__ float errs[];

  errs[threadIdx.x] = y_error[threadIdx.x  + ns - h + 1];
  __syncthreads();

  float val = scaninc_block_add_nowrite<float>(errs);

  if (threadIdx.x == h-1) {
    *MO_fst = val;
  }
}

void bfast_step_7a_run(struct bfast_state *s)
{
  int h = (int)((float)s->n * s->hfrac), N = s->N, m = s->m;
  float *d_y_errors = fget_dev(s,y_errors), *d_MO_fsts = fget_dev(s,MO_fsts);
  int *d_nss = iget_dev(s,nss);

  dim3 grid(m, 1, 1);
  dim3 block(h, 1, 1);
  const size_t shared_size = h * sizeof(float);
  bfast_step_7a<<<grid, block, shared_size>>>(d_y_errors, d_nss, h, N,
                                              d_MO_fsts);
}

BFAST_BEGIN_TEST(bfast_step_7a_test)
  BFAST_BEGIN_INPUTS
  {
    BFAST_VALUE_y_errors, BFAST_VALUE_nss
  }
  BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS { BFAST_VALUE_MO_fsts } BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_7a_run) } BFAST_END_STEPS
BFAST_END_TEST

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 7b: Calculating BOUND
//
// Input:
// Output:
//    BOUND: [N-n]f32

__global__ void bfast_step_7b(float lam,
                              int   n,
                              int   N,
                            float  *BOUND)
{
  // Grid: (1, 1, 1)
  // Block: (N-n, 1, 1)

  if ( threadIdx.x < N-n ) {
    float frac = fdividef(n + 1 + threadIdx.x, n);
    BOUND[threadIdx.x] = lam * ( frac>expf(1.0f) ? sqrtf(logf(frac)) : 1);
  }
}

void bfast_step_7b_run(struct bfast_state *s)
{
  float lam = s->lam;
  int n = s->n, N = s->N;
  float *d_BOUND = fget_dev(s,BOUND);

  dim3 grid(1, 1, 1);
  dim3 block(N-n, 1, 1);
  bfast_step_7b<<<grid, block>>>(lam, n, N, d_BOUND);
}

BFAST_BEGIN_TEST(bfast_step_7b_test)
  BFAST_BEGIN_INPUTS { } BFAST_END_INPUTS
  BFAST_BEGIN_OUTPUTS { BFAST_VALUE_BOUND } BFAST_END_OUTPUTS
  BFAST_BEGIN_STEPS { BFAST_STEP(bfast_step_7b_run) } BFAST_END_STEPS
BFAST_END_TEST
