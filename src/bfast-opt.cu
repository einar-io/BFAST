#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <cassert>
#include "bfast-helpers.cu.h"
//#define INVALID_INDEX (-1)

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// Matrix transposition

template <class ElTp, int T>
__global__ void transpose_tiled_kernel(ElTp* A, ElTp* B,
                                       int heightA, int widthA)
{
  extern __shared__ char sh_mem1[];
  volatile ElTp *tile = (volatile ElTp *)sh_mem1;

  int x = blockIdx.x * T + threadIdx.x;
  int y = blockIdx.y * T + threadIdx.y;

  if (x < widthA && y < heightA) {
    tile[threadIdx.y*(T+1) + threadIdx.x] = A[y*widthA + x];
  }

  __syncthreads();

  x = blockIdx.y * T + threadIdx.x;
  y = blockIdx.x * T + threadIdx.y;

  if (x < heightA && y < widthA) {
    B[y*heightA + x] = tile[threadIdx.x*(T+1) + threadIdx.y];
  }
}

template<class ElTp, int T>
void transpose_tiled ( ElTp*              d_in,
                       ElTp*              d_out,
                       const unsigned int height,
                       const unsigned int width)
{
   // 1. setup block and grid parameters
   unsigned int sh_mem_size = T * (T+1) * sizeof(ElTp);
   int  dimy = (height+T-1) / T;
   int  dimx = (width +T-1) / T;
   dim3 block(T, T, 1);
   dim3 grid (dimx, dimy, 1);

   //2. execute the kernel
   transpose_tiled_kernel<ElTp,T><<<grid, block, sh_mem_size>>>
                                 (d_in, d_out, height, width);
   hipDeviceSynchronize();
}

void transpose(float *d_A, float *d_B, int heightA, int widthA)
{
  transpose_tiled<float, 32>(d_A, d_B, heightA, widthA);
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// For bfast_5, bfast_6, bfast_7a

template <class T>
__device__ inline T scaninc_warp_add(volatile T *in)
{
  const unsigned int idx  = threadIdx.x;
  const unsigned int lane = idx & 31;

  // no synchronization needed inside a WARP,
  //   i.e., SIMD execution
  if (lane >= 1)  { in[idx] = in[idx-1]  + in[idx]; }
  if (lane >= 2)  { in[idx] = in[idx-2]  + in[idx]; }
  if (lane >= 4)  { in[idx] = in[idx-4]  + in[idx]; }
  if (lane >= 8)  { in[idx] = in[idx-8]  + in[idx]; }
  if (lane >= 16) { in[idx] = in[idx-16] + in[idx]; }

  return in[idx];
}

template <class T>
__device__ inline void scaninc_block_add(volatile T *in)
{
  const unsigned int idx    = threadIdx.x;
  const unsigned int lane   = idx &  31;
  const unsigned int warpid = idx >> 5;

  T val = scaninc_warp_add(in);
  __syncthreads();

  if (lane == 31) { in[warpid] = val; }
  __syncthreads();

  if (warpid == 0) scaninc_warp_add(in);
  __syncthreads();

  if (warpid > 0) {
    val = in[warpid-1] + val;
  }

  __syncthreads();
  in[idx] = val;
  __syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 1: Generating X
//
// Output:
//   X: [k2p2][N]f32

__global__ void bfast_step_1(float *X, int k2p2, int N, float f)
{
  int gidy = blockIdx.y * blockDim.y + threadIdx.y;
  int gidx = blockIdx.x * blockDim.x + threadIdx.x;

  if (gidy >= k2p2 || gidx >= N) {
    return;
  }

  int i = gidy;
  int j = gidx + 1;
  float val;
  if (i == 0) { val = 1.0; }
  else if (i == 1) { val = (float)j; }
  else {
    float angle = 2.0 * M_PI * (float)(i / 2) * (float)j / f;
    if (i % 2 == 0) {
      val = __sinf(angle);
    } else {
      val = __cosf(angle);
    }
  }

  X[IDX_2D(gidy, gidx, N)] = val;
}

extern "C" void bfast_step_1_single(float **X, int k2p2, int N, float f)
{
  float *d_X;
  const size_t mem_X = k2p2 * N * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_X, mem_X));
  dim3 block(16, 16, 1);
  dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(k2p2, block.y), 1);
  bfast_step_1<<<grid, block>>>(d_X, k2p2, N, f);

  *X = (float *)malloc(mem_X);
  CUDA_SUCCEED(hipMemcpy(*X, d_X, mem_X, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipFree(d_X));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 2: Calculating Xsqr
//
// Input:
//   X:    [k2p2][N]f32    (only using slice: [k2p2][n])
//   Xt:   [N][k2p2]f32    (only using slice: [n][k2p2])
//   Y:    [m][N]f32       (only using slice: [m][n])
// Output:
//   Xsqr: [m][k2p2][k2p2]f32
//

#define STEP_2_TILE_SIZE 28

__global__ void bfast_step_2(float *Xh, float *Xth, float *Yth, float *Xsqr,
    int N, int n, int k2p2, int m)
{
  // Grid: (CEIL_DIV(m, STEP_2_TILE_SIZE), 1, 1)
  // Block: (k2p2, k2p2, 1)

  if (threadIdx.y >= k2p2 || threadIdx.x >= k2p2) {
    return;
  }

  float accum[STEP_2_TILE_SIZE];
  __shared__ float ysh[STEP_2_TILE_SIZE];

  for (int t = 0; t < STEP_2_TILE_SIZE; t++) {
    accum[t] = 0.0;
  }

  for (int i = 0; i < n; i++) {
    float val = Xh[IDX_2D(threadIdx.y, i, N)]
                  * Xth[IDX_2D(i, threadIdx.x, k2p2)];

    int ysh_idx = IDX_2D(threadIdx.y, threadIdx.x, k2p2);
    if (ysh_idx < STEP_2_TILE_SIZE) {
      int y_row = blockIdx.x * STEP_2_TILE_SIZE + ysh_idx;
      if (y_row < m) {
        ysh[ysh_idx] = Yth[IDX_2D(i, y_row, N)];
      } else {
        ysh[ysh_idx] = 0.0;
      }
    }
    __syncthreads();

    for (int t = 0; t < STEP_2_TILE_SIZE; t++) {
      if (!isnan(ysh[t])) {
        accum[t] += val;
      }
    }
  }

  for (int t = 0; t < STEP_2_TILE_SIZE; t++) {
    int mat_idx = blockIdx.x * STEP_2_TILE_SIZE + t;
    if (mat_idx < m) {
      Xsqr[mat_idx * k2p2 * k2p2 + IDX_2D(threadIdx.y, threadIdx.x, k2p2)]
        = accum[t];
    }
  }
}

extern "C" void bfast_step_2_single(float *X, float *Xt, float *Y,
    float **Xsqr, int N, int n, int k2p2, int m)
{
  // XXX: This function should actually take Yt as input, not Y!
  float *d_X = NULL, *d_Xt = NULL, *d_Y = NULL, *d_Yt = NULL, *d_Xsqr = NULL;
  const size_t mem_X = k2p2 * N * sizeof(float);
  const size_t mem_Y = m * N * sizeof(float);
  const size_t mem_Xsqr = m * k2p2 * k2p2 * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_X, mem_X));
  CUDA_SUCCEED(hipMalloc(&d_Xt, mem_X));
  CUDA_SUCCEED(hipMalloc(&d_Y, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_Yt, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_Xsqr, mem_Xsqr));

  CUDA_SUCCEED(hipMemcpy(d_X, X, mem_X, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_Xt, Xt, mem_X, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_Y, Y, mem_Y, hipMemcpyHostToDevice));

  transpose(d_Y, d_Yt, m, N);

  dim3 block(8, 8, 1); // Assumes k2p2 <= 8
  dim3 grid(CEIL_DIV(m, STEP_2_TILE_SIZE), 1, 1);
  bfast_step_2<<<grid, block>>>(d_X, d_Xt, d_Yt, d_Xsqr, N, n, k2p2, m);

  *Xsqr = (float *)malloc(mem_Xsqr);
  CUDA_SUCCEED(hipMemcpy(*Xsqr, d_Xsqr, mem_Xsqr, hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_X));
  CUDA_SUCCEED(hipFree(d_Xt));
  CUDA_SUCCEED(hipFree(d_Y));
  CUDA_SUCCEED(hipFree(d_Xsqr));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 3: Calculating Xinv
//
// Input:
//   Xsqr: [m][k2p2][k2p2]f32
// Output:
//   Xinv: [m][k2p2][k2p2]f32
//

__global__ void bfast_step_3(float *Xsqr, float *Xinv, int k2p2)
{
  // Grid: (m, 1, 1)
  // Block: (2*k2p2, k2p2, 1)
  // NB! Uses dynamically allocated shared memory: k2p2*2*k2p2 floats per block

  if (threadIdx.x >= 2*k2p2 || threadIdx.y >= k2p2) {
    return;
  }

  float *sqr = &Xsqr[blockIdx.x * (k2p2 * k2p2)];
  float *inv = &Xinv[blockIdx.x * (k2p2 * k2p2)];

  extern __shared__ float A[]; // [k2p2][2*k2p2]

  // Body of mat_inv map
  if (threadIdx.x < k2p2) {
    // Left half of A
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] =
      sqr[IDX_2D(threadIdx.y, threadIdx.x, k2p2)];
  } else {
    // Right half of A
    float val = threadIdx.y == (threadIdx.x - k2p2) ? 1.0 : 0.0;
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] = val;
  }
  __syncthreads();

  // guass_jordan loop and map body
  for (int i = 0; i < k2p2; i++) {
    float v1 = A[i];
    float x = A[threadIdx.x] / v1;
    float val = x;

    if (threadIdx.y < k2p2 - 1) {
      val = A[IDX_2D(threadIdx.y + 1, threadIdx.x, 2*k2p2)]
              - A[IDX_2D(threadIdx.y + 1, i, 2*k2p2)] * x;
    }
    __syncthreads();
    A[IDX_2D(threadIdx.y, threadIdx.x, 2*k2p2)] = val;
    __syncthreads();
  }

  // Write back to global memory
  if (threadIdx.x < k2p2) {
    inv[IDX_2D(threadIdx.y, threadIdx.x, k2p2)] =
      A[IDX_2D(threadIdx.y, threadIdx.x + k2p2, 2*k2p2)];
  }
}

extern "C" void bfast_step_3_single(float *Xsqr, float **Xinv, int k2p2, int m)
{
  float *d_Xsqr = NULL, *d_Xinv = NULL;
  const size_t mem_Xsqr = m * k2p2 * k2p2 * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_Xsqr, mem_Xsqr));
  CUDA_SUCCEED(hipMalloc(&d_Xinv, mem_Xsqr));

  CUDA_SUCCEED(hipMemcpy(d_Xsqr, Xsqr, mem_Xsqr, hipMemcpyHostToDevice));

  dim3 block(16, 8, 1); // Assumes k2p2 <= 8
  dim3 grid(m, 1, 1);
  const size_t shared_size = k2p2 * 2 * k2p2 * sizeof(float);
  bfast_step_3<<<grid, block, shared_size>>>(d_Xsqr, d_Xinv, k2p2);

  *Xinv = (float *)malloc(mem_Xsqr);
  CUDA_SUCCEED(hipMemcpy(*Xinv, d_Xinv, mem_Xsqr, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipFree(d_Xinv));
  CUDA_SUCCEED(hipFree(d_Xsqr));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 4a: Calculating beta0
//
// Input:
//   X:     [k2p2][N]f32    (only using slice: [k2p2][n])
//   Y:     [m][N]f32       (only using slice: [m][n])
// Output:
//   beta0: [m][k2p2]
//
// This calculation is performed by transposing Y (so its dimensions become
// [N][m]) and then applying (filtered) matrix-matrix multiplication.
// The output will need to be transposed again, since:
//      [k2p2][N] multiplied with [N][m] is [k2p2][m]
//

__global__ void bfast_step_4a(float *Xh, float *Yth, float *beta0t, int k2p2,
    int n, int m, int N)
{
  int gidy = blockIdx.y * blockDim.y + threadIdx.y;
  int gidx = blockIdx.x * blockDim.x + threadIdx.x;

  if(gidy >= k2p2 || gidx >= m) {
    return;
  }

  float accum = 0.0;
  for(int k = 0; k < n; k ++) {
    float val = Yth[IDX_2D(k, gidx, m)];
    if (!isnan(val)) {
      accum += Xh[IDX_2D(gidy, k, N)] * val;
    }
  }

  beta0t[IDX_2D(gidy, gidx, m)] = accum;
}

extern "C" void bfast_step_4a_single(float *X, float *Y, float **beta0,
    int k2p2, int n, int m, int N)
{
  float *d_X = NULL, *d_Y = NULL, *d_Yt = NULL;
  float *d_beta0 = NULL, *d_beta0t = NULL;
  const size_t mem_X = k2p2 * N * sizeof(float);
  const size_t mem_Y = m * N * sizeof(float);
  const size_t mem_beta0 = m * k2p2 * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_X, mem_X));
  CUDA_SUCCEED(hipMalloc(&d_Y, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_Yt, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_beta0, mem_beta0));
  CUDA_SUCCEED(hipMalloc(&d_beta0t, mem_beta0));

  CUDA_SUCCEED(hipMemcpy(d_X, X, mem_X, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_Y, Y, mem_Y, hipMemcpyHostToDevice));

  transpose(d_Y, d_Yt, m, N);

  dim3 block(16, 16, 1);
  dim3 grid(CEIL_DIV(m, block.x), CEIL_DIV(k2p2, block.y), 1);
  bfast_step_4a<<<grid, block>>>(d_X, d_Yt, d_beta0t, k2p2, n, m, N);

  transpose(d_beta0t, d_beta0, k2p2, m);

  *beta0 = (float *)malloc(mem_beta0);
  CUDA_SUCCEED(hipMemcpy(*beta0, d_beta0, mem_beta0, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipFree(d_X));
  CUDA_SUCCEED(hipFree(d_Y));
  CUDA_SUCCEED(hipFree(d_Yt));
  CUDA_SUCCEED(hipFree(d_beta0));
  CUDA_SUCCEED(hipFree(d_beta0t));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 4b: Calculating beta
//
// Input:
//   Xinv:  [m][k2p2][k2p2]f32
//   beta0: [m][k2p2]f32
// Output:
//   beta:  [m][k2p2]f32
//
// Every block performs a matrix-vector multiplication between a matrix from
// Xinv and a row from beta0. The resulting vectors are rows in the final
// [m][k2p2] matrix, beta.
//

__global__ void bfast_step_4b(float *Xinv, float *beta0, float *beta, int k2p2)
{
  // Grid: (m, 1, 1)
  // Block: (k2p2, 1, 1)

  if (threadIdx.x >= k2p2) { return; }

  float *inv = &Xinv[blockIdx.x * (k2p2 * k2p2)];
  float *vct = &beta0[blockIdx.x * k2p2];
  float accum = 0.0;

  for (int i = 0; i < k2p2; i++) {
    accum += inv[IDX_2D(threadIdx.x, i, k2p2)] * vct[i];
  }

  beta[IDX_2D(blockIdx.x, threadIdx.x, blockDim.x)] = accum;
}

extern "C" void bfast_step_4b_single(float *Xinv, float *beta0, float **beta,
    int m, int k2p2)
{
  float *d_Xinv = NULL, *d_beta0 = NULL, *d_beta = NULL;
  const size_t mem_Xinv = m * k2p2 * k2p2 * sizeof(float);
  const size_t mem_beta0 = m * k2p2 * sizeof(float);
  const size_t mem_beta = mem_beta0;

  CUDA_SUCCEED(hipMalloc(&d_Xinv, mem_Xinv));
  CUDA_SUCCEED(hipMalloc(&d_beta0, mem_beta0));
  CUDA_SUCCEED(hipMalloc(&d_beta, mem_beta));

  CUDA_SUCCEED(hipMemcpy(d_Xinv, Xinv, mem_Xinv, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_beta0, beta0, mem_beta0, hipMemcpyHostToDevice));

  dim3 block(8, 1, 1); // Assumes k2p2 <= 8
  dim3 grid(m, 1, 1);
  bfast_step_4b<<<grid, block>>>(d_Xinv, d_beta0, d_beta, k2p2);

  *beta = (float *)malloc(mem_beta);
  CUDA_SUCCEED(hipMemcpy(*beta, d_beta, mem_beta, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipFree(d_Xinv));
  CUDA_SUCCEED(hipFree(d_beta0));
  CUDA_SUCCEED(hipFree(d_beta));
}


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 4c: Calculating y_preds
//
// Input:
//   X:       [k2p2][N]f32
//   beta:    [m][k2p2]f32
// Output:
//   y_preds: [m][N]f32
//
// Similar reasoning as in 4a. Consider merging these two kernels, the only
// difference is the filtering.
//
// Perform matrix-matrix multiplication between X and beta.

__global__ void bfast_step_4c(float *X, float *beta, float *y_preds,
    int N, int m, int k2p2)
{
  int gidy = blockIdx.y * blockDim.y + threadIdx.y;
  int gidx = blockIdx.x * blockDim.x + threadIdx.x;

  if(gidy >= m || gidx >= N) {
    return;
  }

  float accum = 0.0;
  for(int k = 0; k < k2p2; k ++) {
    accum += beta[IDX_2D(gidy, k, k2p2)] * X[IDX_2D(k, gidx, N)];
  }

  y_preds[IDX_2D(gidy, gidx, N)] = accum;
}

extern "C" void bfast_step_4c_single(float *Xt, float *beta, float **y_preds,
    int m, int N, int k2p2)
{
  // XXX: This function should actually take X as input, not Xt!
  float *d_Xt = NULL, *d_beta = NULL, *d_X = NULL;
  float *d_y_preds = NULL;
  const size_t mem_Xt = N * k2p2 * sizeof(float);
  const size_t mem_beta = m * k2p2 * sizeof(float);
  const size_t mem_y_preds = m * N * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_X, mem_Xt));
  CUDA_SUCCEED(hipMalloc(&d_Xt, mem_Xt));
  CUDA_SUCCEED(hipMalloc(&d_beta, mem_beta));
  CUDA_SUCCEED(hipMalloc(&d_y_preds, mem_y_preds));

  CUDA_SUCCEED(hipMemcpy(d_Xt, Xt, mem_Xt, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_beta, beta, mem_beta, hipMemcpyHostToDevice));

  transpose(d_Xt, d_X, N, k2p2);

  dim3 block(16, 16, 1);
  dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(m, block.y), 1);
  bfast_step_4c<<<grid, block>>>(d_X, d_beta, d_y_preds, N, m, k2p2);


  *y_preds = (float *)malloc(mem_y_preds);
  CUDA_SUCCEED(hipMemcpy(*y_preds, d_y_preds, mem_y_preds,
        hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipFree(d_X));
  CUDA_SUCCEED(hipFree(d_Xt));
  CUDA_SUCCEED(hipFree(d_beta));
  CUDA_SUCCEED(hipFree(d_y_preds));
}


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 5: Calculating Nss, y_erros, val_indss
//
// Input:
//   Y:         [m][N]f32
//   y_preds:   [m][N]f32
// Output:
//   Nss:       [m]i32
//   y_errors:  [m][N]f32
//   val_indss: [m][N]i32
//

__global__ void bfast_step_5(float *Y, float *y_preds, int *Nss,
    float *y_errors, int *val_indss, int N)
{
  // Grid: (m, 1, 1)
  // Block: (1024, 1, 1)

  if (threadIdx.x >= N) { return; }

  float *y = &Y[blockIdx.x * N];
  float *y_pred = &y_preds[blockIdx.x * N];
  float *y_error = &y_errors[blockIdx.x * N];
  int *val_inds = &val_indss[blockIdx.x * N];
  int *Ns = &Nss[blockIdx.x];

  float val = y[threadIdx.x];
  float err = !isnan(val) ? val - y_pred[threadIdx.x] : NAN;

  // Partition
  __shared__ int num_valids[1024];
  num_valids[threadIdx.x] = !isnan(err);
  __syncthreads();
  scaninc_block_add<int>(num_valids);
  int i = num_valids[N - 1];

  unsigned int idx;
  if (!isnan(err)) {
    idx = num_valids[threadIdx.x] - 1;
  } else {
    float num_invalids = threadIdx.x - (num_valids[threadIdx.x] - 1);
    idx = num_invalids - 1 + i;
    //idx = threadIdx.x - num_valids[threadIdx.x] + i;
  }

  y_error[idx] = err;
  val_inds[idx] = threadIdx.x;
  if (threadIdx.x == 0) {
    *Ns = i;
  }
}

extern "C" void bfast_step_5_single(float *Y, float *y_preds, int **Nss,
    float **y_errors, int **val_indss, int N, int m)
{
  float *d_Y = NULL, *d_y_preds = NULL, *d_y_errors = NULL;
  int *d_Nss = NULL, *d_val_indss = NULL;
  const size_t mem_Y = m * N * sizeof(float);
  const size_t mem_y_preds = mem_Y;
  const size_t mem_Nss = m * sizeof(float);
  const size_t mem_y_errors = mem_Y;
  const size_t mem_val_indss = mem_Y;

  CUDA_SUCCEED(hipMalloc(&d_Y, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_y_preds, mem_y_preds));
  CUDA_SUCCEED(hipMalloc(&d_Nss, mem_Nss));
  CUDA_SUCCEED(hipMalloc(&d_y_errors, mem_y_errors));
  CUDA_SUCCEED(hipMalloc(&d_val_indss, mem_val_indss));

  CUDA_SUCCEED(hipMemcpy(d_Y, Y, mem_Y, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_y_preds, y_preds, mem_y_preds, hipMemcpyHostToDevice));

  dim3 block(1024, 1, 1);
  dim3 grid(m, 1, 1);
  bfast_step_5<<<grid, block>>>(d_Y, d_y_preds, d_Nss, d_y_errors, d_val_indss, N);

  *Nss = (int *)malloc(mem_Nss);
  *y_errors = (float *)malloc(mem_y_errors);
  *val_indss = (int *)malloc(mem_val_indss);
  CUDA_SUCCEED(hipMemcpy(*Nss, d_Nss, mem_Nss, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipMemcpy(*y_errors, d_y_errors, mem_y_errors,
        hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipMemcpy(*val_indss, d_val_indss, mem_val_indss,
        hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_Y));
  CUDA_SUCCEED(hipFree(d_y_preds));
  CUDA_SUCCEED(hipFree(d_Nss));
  CUDA_SUCCEED(hipFree(d_y_errors));
  CUDA_SUCCEED(hipFree(d_val_indss));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 6: Calculating nss, sigmas
//
// Input:
//   Y:       [m][N]f32     (only using slice: [m][n])
//   y_preds: [m][N]f32
// Output:
//   nss:     [m]i32
//   sigmas:  [m]f32
//

__global__ void bfast_step_6(float *Yh, float *y_errors, int *nss,
    float *sigmas, int n, int N, int k2p2)
{
  // Grid dimensions (x, y, z): (m, 1, 1)
  // Block dimensions (x, y, z ): (1024, 1, 1)

  if (threadIdx.x >= n) { return; }

  float *yh = &Yh[blockIdx.x * N]; // Yh is Y, so N cols in memory
  float *y_error = &y_errors[blockIdx.x * N];

  __shared__ int num_valids[1024];
  num_valids[threadIdx.x] = !isnan(yh[threadIdx.x]);
  __syncthreads();
  scaninc_block_add<int>(num_valids);
  int ns = num_valids[n - 1];

  // hacky optimization: reuse num_valids by ptr cast
  // __shared__ float sigma_shared[1024]; 
  float *sigma_shared = (float *) &num_valids;
  float val = threadIdx.x < ns ? y_error[threadIdx.x] : 0.0;
  val = val * val;
  sigma_shared[threadIdx.x] = val;
  __syncthreads();
  scaninc_block_add<float>(sigma_shared);

  if (threadIdx.x == 0) {
    //float sigma0 = sigma_shared[n - 1];
    //float sigma = sqrtf(sigma0 / ((float)(ns - k2p2)));
    sigmas[blockIdx.x] = __fsqrt_rd(sigma_shared[n - 1] / ((float)(ns - k2p2)));
    nss[blockIdx.x] = ns;
  }
}

extern "C" void bfast_step_6_single(float *Y, float *y_errors,  int **nss,
    float **sigmas, int n, int k2p2, int m, int N)
{
  float *d_Y = NULL, *d_y_errors = NULL, *d_sigmas = NULL;
  int *d_nss = NULL;
  const size_t mem_Y = m * N * sizeof(float);
  const size_t mem_y_errors = mem_Y;
  const size_t mem_nss = m * sizeof(float);
  const size_t mem_sigmas = mem_nss;

  CUDA_SUCCEED(hipMalloc(&d_Y, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_y_errors, mem_y_errors));
  CUDA_SUCCEED(hipMalloc(&d_nss, mem_nss));
  CUDA_SUCCEED(hipMalloc(&d_sigmas, mem_sigmas));

  CUDA_SUCCEED(hipMemcpy(d_Y, Y, mem_Y, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_y_errors, y_errors, mem_y_errors,
        hipMemcpyHostToDevice));

  fprintf(stderr, "n=%d, k2p2=%d, m=%d, N=%d\n", n, k2p2, m, N);
  dim3 block(1024, 1, 1);
  dim3 grid(m, 1, 1);
  bfast_step_6<<<grid, block>>>(d_Y, d_y_errors, d_nss, d_sigmas, n, N, k2p2);

  *nss = (int *)malloc(mem_nss);
  *sigmas = (float *)malloc(mem_sigmas);

  CUDA_SUCCEED(hipMemcpy(*nss, d_nss, mem_nss, hipMemcpyDeviceToHost));
  CUDA_SUCCEED(hipMemcpy(*sigmas, d_sigmas, mem_sigmas,
        hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_Y));
  CUDA_SUCCEED(hipFree(d_y_errors));
  CUDA_SUCCEED(hipFree(d_nss));
  CUDA_SUCCEED(hipFree(d_sigmas));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 7a: Produces some interesting value.
//
// Input:
//    y_errors: [m][N]
//    nss:      [m]
// Output:
//    MO_fsts:  [m]

__global__ void bfast_step_7a(float *y_errors,
                                int *nss,
                                int  h,
                                int  N,
                              float *MO_fsts)
{
  // Grid:  (m, 1, 1)
  // Block: (1024, 1, 1)

  if (h <= threadIdx.x) { return; }

  float *y_error = &y_errors[blockIdx.x * N];
  float *MO_fst  = &MO_fsts [blockIdx.x];
  int    ns      = nss      [blockIdx.x];

  __shared__ float errs[1024];

  errs[threadIdx.x] = y_error[threadIdx.x  + ns - h + 1];
  __syncthreads();

  scaninc_block_add(errs);

  if (threadIdx.x == 0) {
    *MO_fst = errs[h-1];
  }
}

extern "C" void 
bfast_step_7a_single(float  *y_errors,
                       int  *nss,
                       int   h,
                       int   N,
                       int   m,
                     float **MO_fsts)
{
  float *d_y_errors = NULL;
  int   *d_nss      = NULL;
  float *d_MO_fsts  = NULL;

  const size_t mem_y_errors = m * N * sizeof(float);
  const size_t mem_nss      = m * sizeof(float);
  const size_t mem_MO_fsts  = m * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_y_errors, mem_y_errors));
  CUDA_SUCCEED(hipMalloc(&d_nss, mem_nss));
  CUDA_SUCCEED(hipMalloc(&d_MO_fsts, mem_MO_fsts));

  CUDA_SUCCEED(hipMemcpy(d_y_errors, y_errors, mem_y_errors, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_nss, nss, mem_nss, hipMemcpyHostToDevice));

  fprintf(stderr, "h=%d, N=%d, m=%d", h, N, m);

  dim3 grid(m, 1, 1);
  dim3 block(1024, 1, 1);
  bfast_step_7a<<<grid, block>>>(d_y_errors, d_nss, h, N, d_MO_fsts);

  *MO_fsts = (float *)malloc(mem_MO_fsts);
  CUDA_SUCCEED(hipMemcpy(*MO_fsts, d_MO_fsts, mem_MO_fsts, hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_MO_fsts));
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 7b: Produces a BOUND value of at least lam for each step in the monitor period. 
//
// Input:
//    lam:   0
//    n:     0
//    N:     0
// Output:
//    BOUND: [N-n]

__global__ void bfast_step_7b(float lam,
                              int   n,
                              int   N,
                            float  *BOUND)
{
  // Grid: (1, 1, 1)
  // Block: (1024, 1, 1)

  // int monitor_period_sz = N-n;
  if ( threadIdx.x < N-n ) {

    // Index into monitor period
    //unsigned int t = n + 1 + threadIdx.x;

    float frac = fdividef(n + 1 + threadIdx.x, n);

    /*
    // logplus(frac). Assures `tmp` is at least 1.
    if (frac > __expf(1.0f)) { BOUND[threadIdx.x] = lam * __fsqrt_rd( __logf(frac)); }
    else                     { BOUND[threadIdx.x] = lam; }
    */

    //BOUND[threadIdx.x] = lam * ( frac>__expf(1.0f) ? __fsqrt_rd(__logf(frac)) : 1);
    BOUND[threadIdx.x] = lam * ( frac>expf(1.0f) ? sqrtf(logf(frac)) : 1);

  }
}

extern "C" void bfast_step_7b_single(float lam, int n, int N, float
    **BOUND)
{
  float *d_BOUND = NULL;

  const size_t mem_BOUND = (N - n)  * sizeof(float);
  
  CUDA_SUCCEED(hipMalloc(&d_BOUND, mem_BOUND));

  CUDA_SUCCEED(hipMemcpy(d_BOUND, BOUND, mem_BOUND, hipMemcpyHostToDevice));

  fprintf(stderr, "lam=%f, n=%d, N=%d\n", lam, n, N);

  dim3 grid(1, 1, 1);
  dim3 block(1024, 1, 1);
  bfast_step_7b<<<grid, block>>>(lam, n, N, d_BOUND);

  *BOUND = (float *)malloc(mem_BOUND);

  CUDA_SUCCEED(hipMemcpy(*BOUND, d_BOUND, mem_BOUND, hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_BOUND));
}


////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
//  Step 8: Mapping each sample to its excess value w.r.t. its bound. Maybe.
//
// Input:
//   y_errors[]:  [m][N]
//   val_indss[]: [m][N]
//   Nss[]:       [m]
//   nss[]:       [m]
//   sigmas[]:    [m]
//   MO_fsts[]:   [m]
//   BOUND[]:     [N-n]
//   h:
//   m:
//   N:
// Output:
//   breakss[]:   [m][N-n]

__global__ void bfast_step_8(float *y_errors,  // [m][N]
                               int *val_indss, // [m][N]
                               int *Nss,       // [m]
                               int *nss,       // [m]
                             float *sigmas,    // [m]
                             float *MO_fsts,   // [m]
                             float *BOUND,     // [N-n]
                               int h,
                               int n,
                               int N,
                             float *breakss)   // [m][N-n] output
{
  // Layout:
  // Grid:  (m, 1, 1)
  // Block: (1024, 1, 1)

  // Optimization opportunities:
  // Done. Read bound into shared memory.
  // Reuse shared memory for MO MOP MOPP
  // Reuse threadIdx.x instead of copying to local variable.

  if (threadIdx.x >= N-n) { return; }

  // In order of appearence
  int   Ns        = Nss       [blockIdx.x];
  int   ns        = nss       [blockIdx.x];
  float sigma     = sigmas    [blockIdx.x];
  float MO_fst    = MO_fsts   [blockIdx.x];
  float *y_error  = &y_errors [blockIdx.x * N];
  int   *val_inds = &val_indss[blockIdx.x * N];
  float *breaks   = &breakss  [blockIdx.x * (N-n)];
  float val;


  __shared__ float BOUND_shr[1024];
  
  if (threadIdx.x < N) {
    BOUND_shr[threadIdx.x] = BOUND[threadIdx.x];
  }

  __shared__ int val_inds_shr[1024];
  
  if (threadIdx.x < N) {
    val_inds_shr[threadIdx.x] = val_inds[threadIdx.x];
  }

  __shared__ float MO_shr[1024];
  {
    if      ( Ns-ns       <= threadIdx.x ) { MO_shr[threadIdx.x] = 0.0f;   }
    else if ( threadIdx.x == 0           ) { MO_shr[threadIdx.x] = MO_fst; }
    else                   { MO_shr[threadIdx.x] = -y_error[ns - h + threadIdx.x] 
                                                  + y_error[ns + threadIdx.x]; }
    __syncthreads();
    scaninc_block_add<float>(MO_shr);
  }

  {
    // MO'
    __syncthreads();
    //MO_shr[threadIdx.x] = fdividef( MO_shr[threadIdx.x] , sigma * __fsqrt_rd( (float)ns ));
    MO_shr[threadIdx.x] = fdividef( MO_shr[threadIdx.x] , sigma ) * rsqrtf( (float)ns );
  }

  {
    __syncthreads();

    if ( threadIdx.x < Ns - ns ) {
      val = MO_shr[threadIdx.x];
    }
    else {
      val = NAN;
    }

    // Make sure all threads has read into `val` before overwriting source.
    __syncthreads();
    MO_shr[val_inds_shr[threadIdx.x + ns] - n] = val;
  }

  // Here might be a producer/consumer dependency in MO_shr.

  {
    // breaks = ..
    __syncthreads();
    float m = MO_shr   [threadIdx.x];
    float b = BOUND_shr[threadIdx.x];

    if (isnan(m) || isnan(b)) { breaks[threadIdx.x] = 0.0f; }
    else                      { breaks[threadIdx.x] = fabsf(m) - b; }
  }

}




extern "C" void
bfast_step_8_single(float  *y_errors,  // [m][N]
                      int  *val_indss, // [m][N]
                      int  *Nss,       // [m]
                      int  *nss,       // [m]
                    float  *sigmas,    // [m]
                    float  *MO_fsts,   // [m]
                    float  *BOUND,     // [N-n]
                      int  h,
                      int  m,
                      int  N,
                      int  n,
                    float **breakss)   // [m][N-n]
{

  float *d_y_errors  = NULL;
  int   *d_val_indss = NULL;
  int   *d_Nss       = NULL;
  int   *d_nss       = NULL;
  float *d_sigmas    = NULL;
  float *d_MO_fsts   = NULL;
  float *d_BOUND     = NULL;
  float *d_breakss = NULL;

  const size_t mem_y_errors  = m * N * sizeof(float);
  const size_t mem_val_indss = m * N * sizeof(int);
  const size_t mem_Nss       = m     * sizeof(int);
  const size_t mem_nss       = m     * sizeof(int);
  const size_t mem_sigmas    = m     * sizeof(float);
  const size_t mem_MO_fsts   = m     * sizeof(float);
  const size_t mem_BOUND     =     N * sizeof(float);

  const size_t mem_breakss   = m * N * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_y_errors,  mem_y_errors));
  CUDA_SUCCEED(hipMalloc(&d_val_indss, mem_val_indss));
  CUDA_SUCCEED(hipMalloc(&d_Nss,       mem_Nss));
  CUDA_SUCCEED(hipMalloc(&d_nss,       mem_nss));
  CUDA_SUCCEED(hipMalloc(&d_sigmas,    mem_sigmas));
  CUDA_SUCCEED(hipMalloc(&d_MO_fsts,   mem_MO_fsts));
  CUDA_SUCCEED(hipMalloc(&d_BOUND,     mem_BOUND));
  CUDA_SUCCEED(hipMalloc(&d_breakss,     mem_breakss));

  CUDA_SUCCEED(hipMemcpy(d_y_errors,  y_errors,  mem_y_errors,  hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_val_indss, val_indss, mem_val_indss, hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_Nss,       Nss,       mem_Nss,       hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_nss,       nss,       mem_nss,       hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_sigmas,    sigmas,    mem_sigmas,    hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_MO_fsts,   MO_fsts,   mem_MO_fsts,   hipMemcpyHostToDevice));
  CUDA_SUCCEED(hipMemcpy(d_BOUND,     BOUND,     mem_BOUND,     hipMemcpyHostToDevice));

  fprintf(stderr, "h=%d, m=%d, n=%d\n", h, m, n);

  dim3 grid(m, 1, 1);
  dim3 block(1024, 1, 1);
  bfast_step_8<<<grid, block>>>
  (d_y_errors, d_val_indss, d_Nss, d_nss, d_sigmas, d_MO_fsts, d_BOUND, h, n, N, d_breakss);

  *breakss = (float *)malloc(mem_breakss);
  CUDA_SUCCEED(hipMemcpy(*breakss, d_breakss, mem_breakss, hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_MO_fsts));
  CUDA_SUCCEED(hipFree(d_y_errors));
  CUDA_SUCCEED(hipFree(d_val_indss));
  CUDA_SUCCEED(hipFree(d_Nss));
  CUDA_SUCCEED(hipFree(d_nss));
  CUDA_SUCCEED(hipFree(d_sigmas));
  CUDA_SUCCEED(hipFree(d_BOUND));
  CUDA_SUCCEED(hipFree(d_breakss));
}

extern "C" void bfast_naive(struct bfast_in *in, struct bfast_out *out)
{
  int k = in->k;
  int n = in->n;
  float f = in->freq;
  float hfrac = in->hfrac;
  float lam = in->lam;
  float *Y = in->images;
  const int m = in->shp[0];
  const int N = in->shp[1];

  int k2p2 = k * 2 + 2;
  int h = (int) ((float)n * hfrac);

  float *d_Y, *d_X, *d_Xt, *d_Xsqr, *d_Xinv, *d_Yt;
  float *d_beta0, *d_beta0t, *d_beta, *d_betat, *d_y_preds, *d_y_predst;
  int *d_Nss, *d_val_indss, *d_nss;
  float *d_sigmas, *d_MO_fsts, *d_y_errors, *d_BOUND, *d_breakss;

  const size_t mem_X = k2p2 * N * sizeof(float);
  const size_t mem_Y = m * N * sizeof(float);
  const size_t mem_Xsqr = m * k2p2 * k2p2 * sizeof(float);
  const size_t mem_Xinv = m * k2p2 * k2p2 * sizeof(float);
  const size_t mem_beta0 = m * k2p2 * sizeof(float);
  const size_t mem_beta = m * k2p2 * sizeof(float);
  const size_t mem_y_preds = m * N * sizeof(float);
  const size_t mem_Nss = m * sizeof(int);
  const size_t mem_y_errors = mem_Y;
  const size_t mem_val_indss = mem_Y;
  const size_t mem_nss = m * sizeof(int);
  const size_t mem_sigmas = mem_nss;
  const size_t mem_MO_fsts = mem_nss;
  const size_t mem_BOUND = (N - n) * sizeof(float);
  const size_t mem_breakss = m * (N - n) * sizeof(float);

  CUDA_SUCCEED(hipMalloc(&d_X, mem_X));
  CUDA_SUCCEED(hipMalloc(&d_Xt, mem_X));
  CUDA_SUCCEED(hipMalloc(&d_Y, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_Xsqr, mem_Xsqr));
  CUDA_SUCCEED(hipMalloc(&d_Xinv, mem_Xinv));
  CUDA_SUCCEED(hipMalloc(&d_Yt, mem_Y));
  CUDA_SUCCEED(hipMalloc(&d_beta0, mem_beta0));
  CUDA_SUCCEED(hipMalloc(&d_beta0t, mem_beta0));
  CUDA_SUCCEED(hipMalloc(&d_beta, mem_beta));
  CUDA_SUCCEED(hipMalloc(&d_betat, mem_beta));
  CUDA_SUCCEED(hipMalloc(&d_y_preds, mem_y_preds));
  CUDA_SUCCEED(hipMalloc(&d_y_predst, mem_y_preds));
  CUDA_SUCCEED(hipMalloc(&d_Nss, mem_Nss));
  CUDA_SUCCEED(hipMalloc(&d_y_errors, mem_y_errors));
  CUDA_SUCCEED(hipMalloc(&d_val_indss, mem_val_indss));
  CUDA_SUCCEED(hipMalloc(&d_nss, mem_nss));
  CUDA_SUCCEED(hipMalloc(&d_sigmas, mem_sigmas));
  CUDA_SUCCEED(hipMalloc(&d_MO_fsts, mem_MO_fsts));
  CUDA_SUCCEED(hipMalloc(&d_BOUND, mem_BOUND));
  CUDA_SUCCEED(hipMalloc(&d_breakss, mem_breakss));

  CUDA_SUCCEED(hipMemcpy(d_Y, Y, mem_Y, hipMemcpyHostToDevice));

  CUDA_SUCCEED(hipDeviceSynchronize());

  struct timer bfast_timer;
  struct timer kernel_timer[11];
  timer_reset(&bfast_timer);
  for (int i = 0; i < sizeof(kernel_timer)/sizeof(kernel_timer[0]); i++) {
    timer_reset(&kernel_timer[i]);
  }

  for (int i = 0; i < num_runs; i++) {
    if (!print_individual) { timer_start(&bfast_timer); }

    {
      timer_individual_start(kernel_timer, 0);
      dim3 block(16, 16, 1);
      dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(k2p2, block.y), 1);
      bfast_step_1<<<grid, block>>>(d_X, k2p2, N, f);
      timer_individual_stop(kernel_timer, 0);
    }

    {
      timer_individual_start(kernel_timer, 1);
      transpose(d_X, d_Xt, k2p2, N);
      transpose(d_Y, d_Yt, m, N);
      dim3 block(8, 8, 1); // Assumes k2p2 <= 8
      dim3 grid(CEIL_DIV(m, STEP_2_TILE_SIZE), 1, 1);
      bfast_step_2<<<grid, block>>>(d_X, d_Xt, d_Yt, d_Xsqr, N, n, k2p2, m);
      timer_individual_stop(kernel_timer, 1);
    }

    {
      timer_individual_start(kernel_timer, 2);
      dim3 block(16, 8, 1); // Assumes k2p2 <= 8
      dim3 grid(m, 1, 1);
      const size_t shared_size = k2p2 * 2 * k2p2 * sizeof(float);
      bfast_step_3<<<grid, block, shared_size>>>(d_Xsqr, d_Xinv, k2p2);
      timer_individual_stop(kernel_timer, 2);
    }

    {
      timer_individual_start(kernel_timer, 3);
      dim3 block(16, 16, 1);
      dim3 grid(CEIL_DIV(m, block.x), CEIL_DIV(k2p2, block.y), 1);
      bfast_step_4a<<<grid, block>>>(d_X, d_Yt, d_beta0t, k2p2, n, m, N);
      transpose(d_beta0t, d_beta0, k2p2, m);
      timer_individual_stop(kernel_timer, 3);
    }

    {
      timer_individual_start(kernel_timer, 4);
      dim3 block(8, 1, 1); // Assumes k2p2 <= 8
      dim3 grid(m, 1, 1);
      bfast_step_4b<<<grid, block>>>(d_Xinv, d_beta0, d_beta, k2p2);
      timer_individual_stop(kernel_timer, 4);
    }

    {
      timer_individual_start(kernel_timer, 5);
      dim3 block(16, 16, 1);
      dim3 grid(CEIL_DIV(N, block.x), CEIL_DIV(m, block.y), 1);
      bfast_step_4c<<<grid, block>>>(d_X, d_beta, d_y_preds, N, m, k2p2);
      timer_individual_stop(kernel_timer, 5);
    }

    {
      timer_individual_start(kernel_timer, 6);
      dim3 block(1024, 1, 1);
      dim3 grid(m, 1, 1);
      bfast_step_5<<<grid, block>>>(d_Y, d_y_preds, d_Nss, d_y_errors, d_val_indss, N);
      timer_individual_stop(kernel_timer, 6);
    }

    {
      timer_individual_start(kernel_timer, 7);
      dim3 block(1024, 1, 1);
      dim3 grid(m, 1, 1);
      bfast_step_6<<<grid, block>>>(d_Y, d_y_errors, d_nss, d_sigmas, n, N, k2p2);
      timer_individual_stop(kernel_timer, 7);
    }

    {
      timer_individual_start(kernel_timer, 8);
      dim3 block(1024, 1, 1);
      dim3 grid(m, 1, 1);
      bfast_step_7a<<<grid, block>>>(d_y_errors, d_nss, h, N, d_MO_fsts);
      timer_individual_stop(kernel_timer, 8);
    }

    {
      timer_individual_start(kernel_timer, 9);
      dim3 block(1024, 1, 1);
      dim3 grid(1, 1, 1);
      bfast_step_7b<<<grid, block>>>(lam, n, N, d_BOUND);
      timer_individual_stop(kernel_timer, 9);
    }

    {
      timer_individual_start(kernel_timer, 10);
      dim3 block(1024, 1, 1);
      dim3 grid(m, 1, 1);
      bfast_step_8<<<grid, block>>>(d_y_errors, d_val_indss, d_Nss, d_nss,
          d_sigmas, d_MO_fsts, d_BOUND, h, n, N, d_breakss);
      timer_individual_stop(kernel_timer, 10);
    }

    if (!print_individual) { timer_stop(&bfast_timer); }
  }

  if (print_individual) {
    for (int i = 0; i < sizeof(kernel_timer)/sizeof(kernel_timer[0]); i++) {
      const char *kernel_name;
      switch (i) {
      case 0:   kernel_name = "bfast_step_1";  break;
      case 1:   kernel_name = "bfast_step_2";  break;
      case 2:   kernel_name = "bfast_step_3";  break;
      case 3:   kernel_name = "bfast_step_4a"; break;
      case 4:   kernel_name = "bfast_step_4b"; break;
      case 5:   kernel_name = "bfast_step_4c"; break;
      case 6:   kernel_name = "bfast_step_5";  break;
      case 7:   kernel_name = "bfast_step_6";  break;
      case 8:   kernel_name = "bfast_step_7a"; break;
      case 9:   kernel_name = "bfast_step_7b"; break;
      case 10:  kernel_name = "bfast_step_8";  break;
      default:  assert(0);
      }
      timer_report(&kernel_timer[i], kernel_name);
    }
  } else {
    timer_report(&bfast_timer, "bfast");
  }

  out->breakss = (float *)malloc(m * (N - n) * sizeof(float));
  out->breakss[0] = 0.0;
  out->shp[0] = m;
  out->shp[1] = N - n;
  CUDA_SUCCEED(hipMemcpy(out->breakss, d_breakss, mem_breakss, hipMemcpyDeviceToHost));

  CUDA_SUCCEED(hipFree(d_X));
  CUDA_SUCCEED(hipFree(d_Xt));
  CUDA_SUCCEED(hipFree(d_Y));
  CUDA_SUCCEED(hipFree(d_Xsqr));
  CUDA_SUCCEED(hipFree(d_Xinv));
  CUDA_SUCCEED(hipFree(d_Yt));
  CUDA_SUCCEED(hipFree(d_beta0));
  CUDA_SUCCEED(hipFree(d_beta0t));
  CUDA_SUCCEED(hipFree(d_beta));
  CUDA_SUCCEED(hipFree(d_betat));
  CUDA_SUCCEED(hipFree(d_y_preds));
  CUDA_SUCCEED(hipFree(d_y_predst));
  CUDA_SUCCEED(hipFree(d_Nss));
  CUDA_SUCCEED(hipFree(d_y_errors));
  CUDA_SUCCEED(hipFree(d_val_indss));
  CUDA_SUCCEED(hipFree(d_nss));
  CUDA_SUCCEED(hipFree(d_sigmas));
  CUDA_SUCCEED(hipFree(d_MO_fsts));
  CUDA_SUCCEED(hipFree(d_BOUND));
  CUDA_SUCCEED(hipFree(d_breakss));
}
